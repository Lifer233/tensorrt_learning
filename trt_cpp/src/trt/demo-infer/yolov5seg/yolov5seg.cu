#include "hip/hip_runtime.h"
#include<infer/trt-infer.hpp>
#include<common/basic_tools.hpp>
#include<common/cuda-tools.hpp>
#include <common/trt-tensor.hpp>
#include <common/matrix.hpp>
#include <opencv2/opencv.hpp>
#include<demo-infer/yolov5seg/yolov5seg.h>

static const int CLASSES = 80;

using namespace std;
using namespace cv;

Matrix mygemm(const Matrix& a, const Matrix& b){

    Matrix c(a.rows(), b.cols());
    for(int i = 0; i < c.rows(); ++i){
        for(int j = 0; j < c.cols(); ++j){
            float summary = 0;
            for(int k = 0; k < a.cols(); ++k)
                summary += a(i, k) * b(k, j);

            c(i, j) = summary;
        }
    }
    return c;
}

struct yolov5out_seg {
	int id;             
	float confidence;   
	Rect box;       
	Mat boxMask;
};

struct yolov5seg_bbox{
    float left, top, right, bottom, confidence;
    int class_label;
    Rect box;
    Mat boxMask;
    Matrix mask_cofs;
    yolov5seg_bbox() = default;
    vector<float> proto;
    yolov5seg_bbox(float left, float top, float right, float bottom, float confidence, int class_label, Matrix mask_cofs, Rect box, vector<float> proto)
        : left(left), top(top), right(right), bottom(bottom), confidence(confidence), class_label(class_label), mask_cofs(mask_cofs), box(box) ,proto(proto){}
};

void DrawPred(Mat& img,std::vector<yolov5seg_bbox> result) {
	std::vector<Scalar> color;
	srand(time(0));
    for (int i = 0; i < CLASSES; i++)
    {
        int b = rand() % 256;
		int g = rand() % 256;
		int r = rand() % 256;
		color.push_back(Scalar(b, g, r));
    }
    Mat mask = img.clone();
    for (int i = 0; i < result.size(); i++) {
		int left, top;
		left = result[i].box.x;
		top = result[i].box.y;
		int color_num = i;
		rectangle(img, result[i].box,color[result[i].class_label], 2, 8);
        cv::Mat c = mask(result[i].box);

        cv::Mat a = result[i].boxMask;

        c.setTo(color[result[i].class_label], a);
        // imwrite(to_string(i) + "_.png", c);
        std::string label = std::to_string(result[i].class_label) + ":" + std::to_string(result[i].confidence);
        int baseLine;
		Size labelSize = getTextSize(label, FONT_HERSHEY_SIMPLEX, 0.5, 1, &baseLine);
		top = max(top, labelSize.height);
		putText(img, label, Point(left, top), FONT_HERSHEY_SIMPLEX, 1, color[result[i].class_label], 2);
	}
	addWeighted(img, 0.5, mask, 0.5, 0, img);

};



void Yolov5Seg::yolov5Seg_inference(){

    auto engine = TRT::load_infer("/home/rex/Desktop/tensorrt_learning/trt_cpp/workspace/yolov5s-seg.trtmodel");
    if(!engine){
        printf("load engine failed \n");
        return;
    }
    auto input       = engine->input();
    auto output      = engine->output();
    auto output1      = engine->output(1);

    int input_width  = input->width();
    int input_height = input->height();
    auto image = imread("/home/rex/Desktop/tensorrt_learning/trt_cpp/workspace/bus.jpg");
    auto img_o = image.clone();
    int img_w = image.cols;
    int img_h = image.rows;
    Mat input_image;
    resize(image,input_image,Size(640,640));
    Mat show_img = input_image.clone();

    input_image.convertTo(input_image, CV_32F);

    Mat channel_based[3];
    for(int i = 0; i < 3; ++i)
        channel_based[i] = Mat(input_height, input_width, CV_32F, input->cpu<float>(0, 2-i));

    split(input_image, channel_based);
    for(int i = 0; i < 3; ++i)
        channel_based[i] = (channel_based[i] / 255.0f);
    
    engine->forward(true);

    float *prob = output->cpu<float>();
    float *prob1 = output1->cpu<float>();

    float *predict = prob1;
    int cols = 117; // 85 + 32
    int len_proto = 32;
    int num_classes = 80 ;
    int rows = 25200;
    vector<yolov5seg_bbox> boxes;
    float confidence_threshold = 0.3;
    float nms_threshold = 0.5;

    for(int i = 0; i < rows; ++i){
        float* pitem = predict + i * cols;
        float objness = pitem[4];
        if(objness < confidence_threshold)
            continue;

        float* pclass = pitem + 5;
        int label     = std::max_element(pclass, pclass + num_classes) - pclass;

        float prob    = pclass[label];
        float confidence = prob * objness;
        if(confidence < confidence_threshold)
            continue;

        float cx     = pitem[0];
        float cy     = pitem[1];
        float width  = pitem[2];
        float height = pitem[3];


        // 通过反变换恢复到图像尺度
        float left   = (cx - width * 0.5);
        float top    = (cy - height * 0.5);
        float right  = (cx + width * 0.5);
        float bottom = (cy + height * 0.5);
        Rect rect(left,top,width,height);

        vector<float> temp_proto(pitem + 5 + num_classes, pitem + 5 + num_classes + len_proto);
        Matrix tmp_cof(1, 32, temp_proto);
        boxes.emplace_back(left, top, right, bottom, confidence, (float)label,tmp_cof,rect,temp_proto);
        
    }
    std::sort(boxes.begin(), boxes.end(), [](yolov5seg_bbox &a, yolov5seg_bbox &b)
              { return a.confidence > b.confidence; });
    std::vector<bool> remove_flags(boxes.size());
    std::vector<yolov5seg_bbox> box_result;
    box_result.reserve(boxes.size());

    auto iou = [](const yolov5seg_bbox& a, const yolov5seg_bbox& b){
        float cross_left   = std::max(a.left, b.left);
        float cross_top    = std::max(a.top, b.top);
        float cross_right  = std::min(a.right, b.right);
        float cross_bottom = std::min(a.bottom, b.bottom);

        float cross_area = std::max(0.0f, cross_right - cross_left) * std::max(0.0f, cross_bottom - cross_top);
        float union_area = std::max(0.0f, a.right - a.left) * std::max(0.0f, a.bottom - a.top) 
                        + std::max(0.0f, b.right - b.left) * std::max(0.0f, b.bottom - b.top) - cross_area;
        if(cross_area == 0 || union_area == 0) return 0.0f;
        return cross_area / union_area;
    };

    for(int i = 0; i < boxes.size(); ++i){
        if(remove_flags[i]) continue;

        auto& ibox = boxes[i];
        box_result.emplace_back(ibox);
        for(int j = i + 1; j < boxes.size(); ++j){
            if(remove_flags[j]) continue;

            auto& jbox = boxes[j];
            if(ibox.class_label == jbox.class_label){
                // class matched
                if(iou(ibox, jbox) >= nms_threshold)
                    remove_flags[j] = true;
            }
        }
    }
    
    // seg 
	Mat maskProposals;
    vector<float> mask_all;
    for (int i = 0; i < box_result.size(); i++)
    {
        vector<float> tmpP = box_result[i].proto;
        maskProposals.push_back(Mat(tmpP).t());

        for (int k = 0; k < tmpP.size();k++){
            mask_all.push_back(tmpP[k]);
        }
    };
    Matrix mask_cof(box_result.size(), len_proto,mask_all);

    float *pdata = prob;
    int _segChannels = 32;
    int _segWidth = 160;
    int _segHeight = 160;
    int INPUT_H = 640;
    int INPUT_W = 640;
    int MASK_THRESHOLD = 0.5;
	vector<float> mask(pdata, pdata + _segChannels * _segWidth * _segHeight);
    Matrix seg_proto(32, 160 * 160, mask);

    // 5 * (160*160)
    // Matrix reultSeg = mygemm(mask_cof, seg_proto);

    Mat mask_protos = Mat(mask);
    // // reshape 成 32 * 160
    Mat protos = mask_protos.reshape(0, { _segChannels,_segWidth * _segHeight });
	std::cout<<protos.size<<std::endl;
    Mat matmulRes = (maskProposals * protos).t();
    Mat masks = matmulRes.reshape(box_result.size(), { _segWidth,_segHeight });
    std::vector<Mat> maskChannels;
    // 5 个 160 * 160 的矩阵
    split(masks, maskChannels);
    
    for (int i = 0; i < box_result.size(); ++i) {
        cout << 999 << endl;
        
        // 1 * (160*160)
        Matrix cur_seg = (mygemm(box_result[i].mask_cofs,seg_proto).exp(-1) + 1.0).power(-1);
        cout << 888 << endl;

        Mat m_dest(cur_seg.data_);
        m_dest = m_dest.reshape(0,{160,160});

        cout << m_dest.rows << endl;
        cout << m_dest.cols << endl;

        Rect m_roi(0, 0, 160, 160);
        m_dest = m_dest(m_roi);
        Mat m_mask;
        resize(m_dest, m_mask, Size(640,640), INTER_NEAREST);
        
        // cout << 0 << endl;
        
        // Mat dest, mask;
        // //sigmod
        // cv::exp(-maskChannels[i], dest);
        // // dist 160 * 160
        // dest = 1 / (1.0 + dest);

        // Rect roi(0,0,160,160);
		// dest = dest(roi);
        // resize(dest, mask, Size(640,640), INTER_NEAREST);

        Rect temp_rect = box_result[i].box;
        cv::Mat b;
        inRange(m_mask(temp_rect), 0.5, 1, b);

        Point classIdPoint;
        double max_class_socre;
		minMaxLoc(b, 0, &max_class_socre, 0, &classIdPoint);
		max_class_socre = (float)max_class_socre;
		box_result[i].boxMask = b;
        cv::imwrite(to_string(i) + "_.jpg", b);
    }

     for (int i = 0; i < box_result.size(); ++i){
        //取 reultSeg 的每一行
     }


    DrawPred(show_img, box_result);
    cv::imwrite("output-seg.jpg", show_img);
}