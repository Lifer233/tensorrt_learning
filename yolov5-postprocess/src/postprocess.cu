#include "hip/hip_runtime.h"
#include "postprocess.h"


__device__ void affine_project(float* matrix, float x, float y, float* ox, float* oy){
    *ox = matrix[0] * x + matrix[1] * y + matrix[2];
    *oy = matrix[3] * x + matrix[4] * y + matrix[5];
}

__global__ void decode_kernel(
    float* predict, int num_bboxes, int num_classes, float confidence_threshold, 
    float* invert_affine_matrix, float* parray, int max_objects, int NUM_BOX_ELEMENT
){  

    // 每个线程处理一个bbox
    int position = blockDim.x * blockIdx.x + threadIdx.x;

    // 超出最大索引return
    if (position >= num_bboxes) return;

    // 当前bbox起始位置的指针
    float* pitem     = predict + (5 + num_classes) * position;
    float objectness = pitem[4];
    if(objectness < confidence_threshold)
        return;


    // 80个类别的起始位置的指针
    float* class_confidence = pitem + 5;
    float confidence        = *class_confidence++;
    int label               = 0;

    // 遍历获得  最大confidence 以及对应的label
    for(int i = 1; i < num_classes; ++i, ++class_confidence){
        if(*class_confidence > confidence){
            confidence = *class_confidence;
            label      = i;
        }
    }

    confidence *= objectness;
    if(confidence < confidence_threshold)
        return;


    //parray = [count, box1, box2, box3]
    //atomicAdd 的含义是 count++ 并且返回旧的值
    int index = atomicAdd(parray, 1);
    if(index >= max_objects)
        return;

    // 计算坐标
    float cx         = *pitem++;
    float cy         = *pitem++;
    float width      = *pitem++;
    float height     = *pitem++;
    float left   = cx - width * 0.5f;
    float top    = cy - height * 0.5f;
    float right  = cx + width * 0.5f;
    float bottom = cy + height * 0.5f;
    // affine_project(invert_affine_matrix, left,  top,    &left,  &top);
    // affine_project(invert_affine_matrix, right, bottom, &right, &bottom);

    // left, top, right, bottom, confidence, class, keepflag
    // keepflag 用于储存是否需要被删除
    
    // index 表示当前 结果偏离parray box的个数
    // 加1 是因为第一个位置是count
    float* pout_item = parray + 1 + index * NUM_BOX_ELEMENT;
    *pout_item++ = left;
    *pout_item++ = top;
    *pout_item++ = right;
    *pout_item++ = bottom;
    *pout_item++ = confidence;
    *pout_item++ = label;
    *pout_item++ = 1; // 1 = keep, 0 = ignore
}

__device__ float box_iou(
    float aleft, float atop, float aright, float abottom, 
    float bleft, float btop, float bright, float bbottom
){

    float cleft 	= max(aleft, bleft);
    float ctop 		= max(atop, btop);
    float cright 	= min(aright, bright);
    float cbottom 	= min(abottom, bbottom);
    
    float c_area = max(cright - cleft, 0.0f) * max(cbottom - ctop, 0.0f);
    if(c_area == 0.0f)
        return 0.0f;
    
    float a_area = max(0.0f, aright - aleft) * max(0.0f, abottom - atop);
    float b_area = max(0.0f, bright - bleft) * max(0.0f, bbottom - btop);
    return c_area / (a_area + b_area - c_area);
}

__global__ void fast_nms_kernel(float* bboxes, int max_objects, float threshold, int NUM_BOX_ELEMENT){

    int position = (blockDim.x * blockIdx.x + threadIdx.x);
    // 最大bbox的数量
    int count = min((int)*bboxes, max_objects);
    if (position >= count) 
        return;                                                                                                                                    
    
    // left, top, right, bottom, confidence, class, keepflag
    // 输出结果中当前 boxx的起始位置的指针
    float* pcurrent = bboxes + 1 + position * NUM_BOX_ELEMENT;
    for(int i = 0; i < count; ++i){
        // 每个bbox的起始位置的指针
        float* pitem = bboxes + 1 + i * NUM_BOX_ELEMENT;
        if(i == position || pcurrent[5] != pitem[5]) continue;

        //如果之后的bbox的 confidence> 当前bbox的 confidence
        if(pitem[4] >= pcurrent[4]){
            // 两个bboxs属于同一类 keep_flag 默认为1
            if(pitem[4] == pcurrent[4] && i < position)
                continue;

            //同类bbox 利用iou除去 重叠度高的框
            float iou = box_iou(
                pcurrent[0], pcurrent[1], pcurrent[2], pcurrent[3],
                pitem[0],    pitem[1],    pitem[2],    pitem[3]
            );

            if(iou > threshold){
                pcurrent[6] = 0;  // 1=keep, 0=ignore
                return;
            }
        }
    }
} 

void decode_kernel_invoker(
    float* predict, int num_bboxes, int num_classes, float confidence_threshold, 
    float nms_threshold, float* invert_affine_matrix, float* parray, int max_objects, int NUM_BOX_ELEMENT, hipStream_t stream){
    
    auto block = num_bboxes > 512 ? 512 : num_bboxes;
    auto grid = (num_bboxes + block - 1) / block;

    decode_kernel<<<grid, block, 0, stream>>>(
        predict, num_bboxes, num_classes, confidence_threshold, 
        invert_affine_matrix, parray, max_objects, NUM_BOX_ELEMENT
    );

    block = max_objects > 512 ? 512 : max_objects;
    grid = (max_objects + block - 1) / block;
    fast_nms_kernel<<<grid, block, 0, stream>>>(parray, max_objects, nms_threshold, NUM_BOX_ELEMENT);
}



std::vector<Box> cpu_decode(float* predict, int rows, int cols, float confidence_threshold , float nms_threshold ){

    std::vector<Box> boxes;
    // 类别数量
    int num_classes = cols - 5;

    for(int i = 0; i < rows; ++i){
        //每一行的起始位置的指针  
        //每一行 x,y,w,h,socre + 80
        
        float* pitem = predict + i * cols;
        
        // score  前景和背景的概率
        float objness = pitem[4];
        //低于阈值的直接过滤,节约计算时间
        if(objness < confidence_threshold)
            continue;

        //pitem 为一行的起始指针 +5 是从第六个位置开始 指向类别概率的 中的第一个位置
        float* pclass = pitem + 5;
        
        // 从指针pclass 到 pclass + num_classes 最大位置的指针  获得偏移量
        int label     = std::max_element(pclass, pclass + num_classes) - pclass;
        
        // pclass[label] 往后找label个位置 对应的值   
        // 最大类别概率
        float prob    = pclass[label];

        // 置信度
        float confidence = prob * objness;
        if(confidence < confidence_threshold)
            continue;
        //计算坐标 
        float cx     = pitem[0];
        float cy     = pitem[1];
        float width  = pitem[2];
        float height = pitem[3];
        float left   = cx - width * 0.5;
        float top    = cy - height * 0.5;
        float right  = cx + width * 0.5;
        float bottom = cy + height * 0.5;
        boxes.emplace_back(left, top, right, bottom, confidence, (float)label);
    }

    //nms
    // lambda 函数 采用引用 避免拷贝   
    std::sort(boxes.begin(), boxes.end(), [](Box& a, Box& b){return a.confidence > b.confidence;});
    
    // 用true false 储存需要删除的 box
    std::vector<bool> remove_flags(boxes.size());
    std::vector<Box> box_result;
    // 先个 box_result 分配 boxes.size() 固定内存，因为 emplace_back 会使得地址发生变化
    // 而使用reserve 后内存只会变化一次,且会节约内存
    box_result.reserve(boxes.size());

    // 计算两个 box之间的 iou
    auto iou = [](const Box& a, const Box& b){
        float cross_left   = std::max(a.left, b.left);
        float cross_top    = std::max(a.top, b.top);
        float cross_right  = std::min(a.right, b.right);
        float cross_bottom = std::min(a.bottom, b.bottom);

        float cross_area = std::max(0.0f, cross_right - cross_left) * std::max(0.0f, cross_bottom - cross_top);
        float union_area = std::max(0.0f, a.right - a.left) * std::max(0.0f, a.bottom - a.top) 
                         + std::max(0.0f, b.right - b.left) * std::max(0.0f, b.bottom - b.top) - cross_area;
        if(cross_area == 0 || union_area == 0) return 0.0f;
        return cross_area / union_area;
    };

    for(int i = 0; i < boxes.size(); ++i){
        if(remove_flags[i]) continue;

        auto& ibox = boxes[i];
        // nms之后的结果
        box_result.emplace_back(ibox);
        for(int j = i + 1; j < boxes.size(); ++j){
            if(remove_flags[j]) continue;

            auto& jbox = boxes[j];

            // 如果两个box属于同一类才进行比较 可以节约时间
            if(ibox.label == jbox.label){
                // class matched
                if(iou(ibox, jbox) >= nms_threshold)
                    remove_flags[j] = true;
            }
        }
    }
    return box_result;
}

std::vector<Box> gpu_decode(float* predict, int rows, int cols, float confidence_threshold , float nms_threshold ){
    
    std::vector<Box> box_result;
    hipStream_t stream = nullptr;
    checkRuntime(hipStreamCreate(&stream));

    float* predict_device = nullptr;
    float* output_device = nullptr;
    float* output_host = nullptr;
    // 由于在cpu上 是顺序执行的，可以vector emplace_back 
    // 而在GPU上是并行的  可以用[count box1 box2] 储存
    // 假定一张图上只有1000个框，超过1000个框的过滤
    int max_objects = 1000;
    int NUM_BOX_ELEMENT = 7;  // left, top, right, bottom, confidence, class, keepflag
    checkRuntime(hipMalloc(&predict_device, rows * cols * sizeof(float)));
    checkRuntime(hipMalloc(&output_device, sizeof(float) + max_objects * NUM_BOX_ELEMENT * sizeof(float)));
    checkRuntime(hipHostMalloc(&output_host, sizeof(float) + max_objects * NUM_BOX_ELEMENT * sizeof(float)));

    checkRuntime(hipMemcpyAsync(predict_device, predict, rows * cols * sizeof(float), hipMemcpyHostToDevice, stream));
    decode_kernel_invoker(
        predict_device, rows, cols - 5, confidence_threshold, 
        nms_threshold, nullptr, output_device, max_objects, NUM_BOX_ELEMENT, stream
    );
    checkRuntime(hipMemcpyAsync(output_host, output_device, 
        sizeof(int) + max_objects * NUM_BOX_ELEMENT * sizeof(float), 
        hipMemcpyDeviceToHost, stream
    ));
    checkRuntime(hipStreamSynchronize(stream));

    // 所有框的个数
    int num_boxes = min((int)output_host[0], max_objects);
    for(int i = 0; i < num_boxes; ++i){
        float* ptr = output_host + 1 + NUM_BOX_ELEMENT * i;
        int keep_flag = ptr[6];
        if(keep_flag){
            box_result.emplace_back(
                ptr[0], ptr[1], ptr[2], ptr[3], ptr[4], (int)ptr[5]
            );
        }
    }
    checkRuntime(hipStreamDestroy(stream));
    checkRuntime(hipFree(predict_device));
    checkRuntime(hipFree(output_device));
    checkRuntime(hipHostFree(output_host));
    return box_result;
}

std::vector<uint8_t> load_file(const std::string& file){

    std::ifstream in(file, std::ios::in | std::ios::binary);
    if (!in.is_open())
        return {};

    in.seekg(0, std::ios::end);
    size_t length = in.tellg();

    std::vector<uint8_t> data;
    if (length > 0){
        in.seekg(0, std::ios::beg);
        data.resize(length);

        in.read((char*)&data[0], length);
    }
    in.close();
    return data;
}

bool __check_cuda_runtime(hipError_t code, const char* op, const char* file, int line){
    if(code != hipSuccess){
        const char* err_name = hipGetErrorName(code);    
        const char* err_message = hipGetErrorString(code);  
        printf("runtime error %s:%d  %s failed. \n  code = %s, message = %s\n", file, line, op, err_name, err_message);   
        return false;
    }
    return true;
}