#include "hip/hip_runtime.h"
#include <warpaffine.h>

void AffineMatrix::invertAffineTransform(float imat[6], float omat[6]){
        float i00 = imat[0];  float i01 = imat[1];  float i02 = imat[2];
        float i10 = imat[3];  float i11 = imat[4];  float i12 = imat[5];

        float D = i00 * i11 - i01 * i10;
        D = D != 0 ? 1.0 / D : 0;
        
        printf("DDD:%f",D);

        float A11 = i11 * D;
        float A22 = i00 * D;
        float A12 = -i01 * D;
        float A21 = -i10 * D;
        float b1 = -A11 * i02 - A12 * i12;
        float b2 = -A21 * i02 - A22 * i12;
        omat[0] = A11;  omat[1] = A12;  omat[2] = b1;
        omat[3] = A21;  omat[4] = A22;  omat[5] = b2;
};
void  AffineMatrix::compute(const MySize& from, const MySize& to){
    float scale_x = to.width / (float)from.width;
    float scale_y = to.height / (float)from.height;

    float scale = min(scale_x, scale_y); 
    
    //正变换M矩阵

    // M = [ scale,    0,     -scale * from.width * 0.5 + to.width * 0.5
    //     0,     scale,    -scale * from.height * 0.5 + to.height * 0.5
    //     0, 0, 1]
    i2d[0] = scale;  
    i2d[1] = 0;  
    i2d[2] = -scale * from.width  * 0.5  + to.width * 0.5 + scale * 0.5 - 0.5;
    i2d[3] = 0;  
    i2d[4] = scale;  
    i2d[5] = -scale * from.height * 0.5 + to.height * 0.5 + scale * 0.5 - 0.5;
    

    //计算M矩阵的逆变换
    invertAffineTransform(i2d, d2i);
}
;

__device__ void affine_project(float* matrix, int x, int y, float* proj_x, float* proj_y){
    
    *proj_x = matrix[0] * x + matrix[1] * y + matrix[2];
    *proj_y = matrix[3] * x + matrix[4] * y + matrix[5];
}

__global__  void warp_affine_bilinear_kernel(
    uint8_t* src, int src_line_size, int src_width, int src_height, 
    uint8_t* dst, int dst_line_size, int dst_width, int dst_height, 
	uint8_t fill_value, AffineMatrix matrix
){
    // 线程ID的全局索引

    int dx = blockDim.x * blockIdx.x + threadIdx.x; 
    int dy = blockDim.y * blockIdx.y + threadIdx.y;

    // 线程ID 超过图像大小时 return
    if (dx >= dst_width || dy >= dst_height)  return;
    
    // 目标图像为640*640*3 用fill_value填充
    float c0 = fill_value, c1 = fill_value, c2 = fill_value;
    float src_x = 0; float src_y = 0;

    //将目标图上一点映射回原图大小
    affine_project(matrix.d2i, dx, dy, &src_x, &src_y);

    if(src_x < -1 || src_x >= src_width || src_y < -1 || src_y >= src_height){
        // out of range
        // src_x < -1，high_x < 0，超出范围
        // src_x >= -1，high_x >= 0，存在取值
    }else{

        // p1  p2
        //   p
        // p3  p4
        int y_low = floorf(src_y);
        int x_low = floorf(src_x);
        int y_high = y_low + 1;
        int x_high = x_low + 1;

        uint8_t const_values[] = {fill_value, fill_value, fill_value};
        
        //双线性差值，和python版一致
        float ly    = src_y - y_low;
        float lx    = src_x - x_low;
        float hy    = 1 - ly;
        float hx    = 1 - lx;
        float w1    = hy * hx, w2 = hy * lx, w3 = ly * hx, w4 = ly * lx;
        uint8_t* v1 = const_values;
        uint8_t* v2 = const_values;
        uint8_t* v3 = const_values;
        uint8_t* v4 = const_values;
        if(y_low >= 0){
            if (x_low >= 0)
                v1 = src + y_low * src_line_size + x_low * 3;

            if (x_high < src_width)
                v2 = src + y_low * src_line_size + x_high * 3;
        }
        
        if(y_high < src_height){
            if (x_low >= 0)
                v3 = src + y_high * src_line_size + x_low * 3;

            if (x_high < src_width)
                v4 = src + y_high * src_line_size + x_high * 3;
        }
        
        // 该点的像素值
        c0 = floorf(w1 * v1[0] + w2 * v2[0] + w3 * v3[0] + w4 * v4[0] + 0.5f);
        c1 = floorf(w1 * v1[1] + w2 * v2[1] + w3 * v3[1] + w4 * v4[1] + 0.5f);
        c2 = floorf(w1 * v1[2] + w2 * v2[2] + w3 * v3[2] + w4 * v4[2] + 0.5f);
    }

    uint8_t* pdst = dst + dy * dst_line_size + dx * 3;
    pdst[0] = c0; pdst[1] = c1; pdst[2] = c2;

    //BGR -> RGB
    // pdst[2] = c0; pdst[1] = c1; pdst[2] = c0;

    // (p - mean) / std 
    // pdst[0] = (c0 - mean) / std;  pdst[1] = (c1 - mean) / std; pdst[2] = (c2 - mean) / std;
    

}

void warp_affine_bilinear(
    uint8_t* src, int src_line_size, int src_width, int src_height, 
    uint8_t* dst, int dst_line_size, int dst_width, int dst_height, 
	uint8_t fill_value
){
    dim3 block_size(32, 32); 
    dim3 grid_size((dst_width + 31) / 32, (dst_height + 31) / 32);
    AffineMatrix affine;
    // affine 
    affine.compute(MySize(src_width, src_height), MySize(dst_width, dst_height));
    warp_affine_bilinear_kernel<<<grid_size, block_size, 0, nullptr>>>(
        src, src_line_size, src_width, src_height,
        dst, dst_line_size, dst_width, dst_height,
        fill_value, affine
    );
}

cv::Mat warpaffine_to_center_align(const cv::Mat& image, const cv::Size& size){  

    cv::Mat output_image(size, CV_8UC3);
    uint8_t* psrc_device = nullptr;
    uint8_t* pdst_device = nullptr;
    size_t src_size = image.cols * image.rows * 3;
    size_t dst_size = size.width * size.height * 3;

    checkRuntime(hipMalloc(&psrc_device, src_size)); 
    checkRuntime(hipMalloc(&pdst_device, dst_size));
    checkRuntime(hipMemcpy(psrc_device, image.data, src_size, hipMemcpyHostToDevice));
    
    // 在cuda上执行warpaffine
    warp_affine_bilinear(
        psrc_device, image.cols * 3, image.cols, image.rows,
        pdst_device, size.width * 3, size.width, size.height,
        114
    );
    checkRuntime(hipPeekAtLastError());
    checkRuntime(hipMemcpy(output_image.data, pdst_device, dst_size, hipMemcpyDeviceToHost));
    checkRuntime(hipFree(psrc_device));
    checkRuntime(hipFree(pdst_device));
    return output_image;
}

bool __check_cuda_runtime(hipError_t code, const char* op, const char* file, int line){
    if(code != hipSuccess){
        const char* err_name = hipGetErrorName(code);    
        const char* err_message = hipGetErrorString(code);  
        printf("runtime error %s:%d  %s failed. \n  code = %s, message = %s\n", file, line, op, err_name, err_message);   
        return false;
    }
    return true;
}