#include "hip/hip_runtime.h"
#include "basic_tools.hpp"
#include "cuda_tools.hpp"
#include <unistd.h>
#include <stdio.h>

using namespace std;
using namespace cv;
bool BaiscTools::exists(const string& path){

#ifdef _WIN32
    return ::PathFileExistsA(path.c_str());
#else
    return access(path.c_str(), R_OK) == 0;
#endif
}

vector<string> BaiscTools::load_labels(const char* file){
    vector<string> lines;

    ifstream in(file, ios::in | ios::binary);
    if (!in.is_open()){
        printf("open %d failed.\n", file);
        return lines;
    }
    
    string line;
    while(getline(in, line)){
        lines.push_back(line);
    }
    in.close();
    return lines;
}


void BaiscTools::AffineMatrix::invertAffineTransform(float imat[6], float omat[6]){
        float i00 = imat[0];  float i01 = imat[1];  float i02 = imat[2];
        float i10 = imat[3];  float i11 = imat[4];  float i12 = imat[5];

        float D = i00 * i11 - i01 * i10;
        D = D != 0 ? 1.0 / D : 0;
        
        printf("DDD:%f",D);

        float A11 = i11 * D;
        float A22 = i00 * D;
        float A12 = -i01 * D;
        float A21 = -i10 * D;
        float b1 = -A11 * i02 - A12 * i12;
        float b2 = -A21 * i02 - A22 * i12;
        omat[0] = A11;  omat[1] = A12;  omat[2] = b1;
        omat[3] = A21;  omat[4] = A22;  omat[5] = b2;
};

void  BaiscTools::AffineMatrix::compute(const MySize& from, const MySize& to){
    float scale_x = to.width / (float)from.width;
    float scale_y = to.height / (float)from.height;
    float scale = min(scale_x, scale_y); 
    i2d[0] = scale;  
    i2d[1] = 0;  
    i2d[2] = -scale * from.width  * 0.5  + to.width * 0.5 + scale * 0.5 - 0.5;
    i2d[3] = 0;  
    i2d[4] = scale;  
    i2d[5] = -scale * from.height * 0.5 + to.height * 0.5 + scale * 0.5 - 0.5;
    invertAffineTransform(i2d, d2i);
};

__device__ void BaiscTools::affine_project(float* matrix, int x, int y, float* proj_x, float* proj_y){
    
    *proj_x = matrix[0] * x + matrix[1] * y + matrix[2];
    *proj_y = matrix[3] * x + matrix[4] * y + matrix[5];
}

__global__  void BaiscTools::warp_affine_bilinear_kernel(
    uint8_t* src, int src_line_size, int src_width, int src_height, 
    uint8_t* dst, int dst_line_size, int dst_width, int dst_height, 
	uint8_t fill_value, AffineMatrix matrix
){
    int dx = blockDim.x * blockIdx.x + threadIdx.x; 
    int dy = blockDim.y * blockIdx.y + threadIdx.y;
    if (dx >= dst_width || dy >= dst_height)  return;
    float c0 = fill_value, c1 = fill_value, c2 = fill_value;
    float src_x = 0; float src_y = 0;
    affine_project(matrix.d2i, dx, dy, &src_x, &src_y);

    if(src_x < -1 || src_x >= src_width || src_y < -1 || src_y >= src_height){
    }else{

        // p1  p2
        //   p
        // p3  p4
        int y_low = floorf(src_y);
        int x_low = floorf(src_x);
        int y_high = y_low + 1;
        int x_high = x_low + 1;

        uint8_t const_values[] = {fill_value, fill_value, fill_value};
        
        //双线性差值，和python版一致
        float ly    = src_y - y_low;
        float lx    = src_x - x_low;
        float hy    = 1 - ly;
        float hx    = 1 - lx;
        float w1    = hy * hx, w2 = hy * lx, w3 = ly * hx, w4 = ly * lx;
        uint8_t* v1 = const_values;
        uint8_t* v2 = const_values;
        uint8_t* v3 = const_values;
        uint8_t* v4 = const_values;
        if(y_low >= 0){
            if (x_low >= 0)
                v1 = src + y_low * src_line_size + x_low * 3;

            if (x_high < src_width)
                v2 = src + y_low * src_line_size + x_high * 3;
        }
        
        if(y_high < src_height){
            if (x_low >= 0)
                v3 = src + y_high * src_line_size + x_low * 3;

            if (x_high < src_width)
                v4 = src + y_high * src_line_size + x_high * 3;
        }
        
        // 该点的像素值
        c0 = floorf(w1 * v1[0] + w2 * v2[0] + w3 * v3[0] + w4 * v4[0] + 0.5f);
        c1 = floorf(w1 * v1[1] + w2 * v2[1] + w3 * v3[1] + w4 * v4[1] + 0.5f);
        c2 = floorf(w1 * v1[2] + w2 * v2[2] + w3 * v3[2] + w4 * v4[2] + 0.5f);
    }

    uint8_t* pdst = dst + dy * dst_line_size + dx * 3;
    pdst[0] = c0; pdst[1] = c1; pdst[2] = c2;

    //BGR -> RGB
    // pdst[2] = c0; pdst[1] = c1; pdst[2] = c0;

    // (p - mean) / std 
    // pdst[0] = (c0 - mean) / std;  pdst[1] = (c1 - mean) / std; pdst[2] = (c2 - mean) / std;

}

void BaiscTools::warp_affine_bilinear(
    uint8_t* src, int src_line_size, int src_width, int src_height, 
    uint8_t* dst, int dst_line_size, int dst_width, int dst_height, 
	uint8_t fill_value
){
    dim3 block_size(32, 32); 
    dim3 grid_size((dst_width + 31) / 32, (dst_height + 31) / 32);
    AffineMatrix affine;
    // affine 
    affine.compute(MySize(src_width, src_height), MySize(dst_width, dst_height));
    warp_affine_bilinear_kernel<<<grid_size, block_size, 0, nullptr>>>(
        src, src_line_size, src_width, src_height,
        dst, dst_line_size, dst_width, dst_height,
        fill_value, affine
    );
}

Mat BaiscTools::warpaffine_to_center_align(const Mat& image, const Size& size){  

    Mat output_image(size, CV_8UC3);
    uint8_t* psrc_device = nullptr;
    uint8_t* pdst_device = nullptr;
    size_t src_size = image.cols * image.rows * 3;
    size_t dst_size = size.width * size.height * 3;

    checkCudaRuntime(hipMalloc(&psrc_device, src_size)); 
    checkCudaRuntime(hipMalloc(&pdst_device, dst_size));
    checkCudaRuntime(hipMemcpy(psrc_device, image.data, src_size, hipMemcpyHostToDevice));
    
    // 在cuda上执行warpaffine
    warp_affine_bilinear(
        psrc_device, image.cols * 3, image.cols, image.rows,
        pdst_device, size.width * 3, size.width, size.height,
        114
    );
    checkCudaRuntime(hipPeekAtLastError());
    checkCudaRuntime(hipMemcpy(output_image.data, pdst_device, dst_size, hipMemcpyDeviceToHost));
    checkCudaRuntime(hipFree(psrc_device));
    checkCudaRuntime(hipFree(pdst_device));
    return output_image;
}

